#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "patch_match.cuh"

#define BLOCK_H 16
#define BLOCK_W 16
#define CUDA_THREADS 1024
#define MIN_BLOCKS_PER_SM 4
#define RANDOM_SEED 3407

__device__ float bilinear_interpolate(
    const float* __restrict__ input_ptr,
    const float2 pt,
    const int32_t height,
    const int32_t width) {
    // get the values of the four vertices
    const int32_t hl = clamp(static_cast<int32_t>(pt.y), 0, height - 1);
    const int32_t wl = clamp(static_cast<int32_t>(pt.x), 0, width - 1);
    const float h_diff = pt.y - static_cast<float>(hl);
    const float w_diff = pt.x - static_cast<float>(wl);

    const int32_t hu = (hl == height - 1) ? hl : hl + 1;
    const int32_t wu = (wl == width - 1) ? wl : wl + 1;

    const float v0 = input_ptr[hl * width + wl];
    const float v1 = input_ptr[hl * width + wu];
    const float v2 = input_ptr[hu * width + wl];
    const float v3 = input_ptr[hu * width + wu];

    // bilinear interpolation
    const float ret = lerp(lerp(v0, v1, w_diff), lerp(v2, v3, w_diff), h_diff);

    return ret;
}

__device__ void sort_small(float* __restrict__ d, const int32_t n) {
    int32_t j;
    for (int32_t i = 1; i < n; i++) {
        float tmp = d[i];
        for (j = i; j >= 1 && tmp < d[j - 1]; j--) d[j] = d[j - 1];
        d[j] = tmp;
    }
}

__device__ int32_t find_min_cost_index(const float* __restrict__ costs, const int32_t n) {
    float min_cost = costs[0];
    int32_t min_cost_idx = 0;
    for (int32_t idx = 1; idx < n; ++idx) {
        if (costs[idx] <= min_cost) {
            min_cost = costs[idx];
            min_cost_idx = idx;
        }
    }
    return min_cost_idx;
}

__device__ void set_bit(uint32_t& input, const uint32_t n) { input |= (uint32_t)(1 << n); }

__device__ int32_t is_set(uint32_t input, const uint32_t n) { return (input >> n) & 1; }

__device__ void mat33_dot_vec3(const float mat[9], const float4 vec, float4* result) {
    result->x = mat[0] * vec.x + mat[1] * vec.y + mat[2] * vec.z;
    result->y = mat[3] * vec.x + mat[4] * vec.y + mat[5] * vec.z;
    result->z = mat[6] * vec.x + mat[7] * vec.y + mat[8] * vec.z;
}

__device__ float vec3_dot_vec3(const float4 vec1, const float4 vec2) {
    return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ void normalize_vec3(float4* __restrict__ vec) {
    const float normSquared = vec->x * vec->x + vec->y * vec->y + vec->z * vec->z;
    const float inverse_sqrt = rsqrtf(normSquared);
    vec->x *= inverse_sqrt;
    vec->y *= inverse_sqrt;
    vec->z *= inverse_sqrt;
}

__device__ void transform_PDF_to_CDF(float* __restrict__ probs, const int32_t num_probs) {
    float prob_sum = 0.0f;
    for (int32_t i = 0; i < num_probs; ++i) {
        prob_sum += probs[i];
    }
    const float inv_prob_sum = 1.0f / prob_sum;

    float cum_prob = 0.0f;
    for (int32_t i = 0; i < num_probs; ++i) {
        const float prob = probs[i] * inv_prob_sum;
        cum_prob += prob;
        probs[i] = cum_prob;
    }
}

// get the point position in the camera coordinate
__device__ void get_3D_point(
    const Camera camera, const int2 p, const float depth, float* __restrict__ X) {
    X[0] = depth * (p.x - camera.K[2]) / camera.K[0];
    X[1] = depth * (p.y - camera.K[5]) / camera.K[4];
    X[2] = depth;
}

__device__ float4 get_view_direction(const Camera camera, const int2 p, const float depth) {
    // get the 3d position of the current pixel according to the depth and
    // intrinsic
    float X[3];
    get_3D_point(camera, p, depth, X);
    float norm = sqrt(X[0] * X[0] + X[1] * X[1] + X[2] * X[2]);

    // normalize the position as view direction
    float4 view_direction;
    view_direction.x = X[0] / norm;
    view_direction.y = X[1] / norm;
    view_direction.z = X[2] / norm;
    view_direction.w = 0;
    return view_direction;
}

__device__ float get_distance_to_origin(
    const Camera camera, const int2 p, const float depth, const float4 normal) {
    // get the 3d position of the current pixel according to the depth and
    // intrinsic
    float X[3];
    get_3D_point(camera, p, depth, X);
    // dot product the normal and point to get the distance to the coordinate
    // origin
    return -(normal.x * X[0] + normal.y * X[1] + normal.z * X[2]);
}

__device__ float compute_depth_from_plane_hypothesis(
    const Camera camera, const float4 plane_hypothesis, const int2 p) {
    return -plane_hypothesis.w * camera.K[0] /
           ((p.x - camera.K[2]) * plane_hypothesis.x +
            (camera.K[0] / camera.K[4]) * (p.y - camera.K[5]) * plane_hypothesis.y +
            camera.K[0] * plane_hypothesis.z);
}

__device__ float4 generate_random_normal(
    const Camera camera, const int2 p, hiprandState* __restrict__ rand_state, const float depth) {
    // randomly generate the normal and store in the plane_hypothesis
    float4 plane_hypothesis;
    float q1 = 1.0f;
    float q2 = 1.0f;
    float s = 2.0f;
    while (s >= 1.0f) {
        q1 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
        q2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
        s = q1 * q1 + q2 * q2;
    }
    const float sq = sqrt(1.0f - s);
    plane_hypothesis.x = 2.0f * q1 * sq;
    plane_hypothesis.y = 2.0f * q2 * sq;
    plane_hypothesis.z = 1.0f - 2.0f * s;
    plane_hypothesis.w = 0;

    // get the view direction in the camera coordinate
    float4 view_direction = get_view_direction(camera, p, depth);
    // assert the angle between the normal and the view direction is greater
    // than 90
    float dot_product = plane_hypothesis.x * view_direction.x +
                        plane_hypothesis.y * view_direction.y +
                        plane_hypothesis.z * view_direction.z;
    if (dot_product > 0.0f) {
        plane_hypothesis.x = -plane_hypothesis.x;
        plane_hypothesis.y = -plane_hypothesis.y;
        plane_hypothesis.z = -plane_hypothesis.z;
    }
    normalize_vec3(&plane_hypothesis);
    return plane_hypothesis;
}

__device__ float4 generate_perturbed_normal(
    const Camera camera,
    const int2 p,
    const float4 normal,
    hiprandState* __restrict__ rand_state,
    const float perturbation) {
    float4 view_direction = get_view_direction(camera, p, 1.0f);

    const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

    const float sin_a1 = sin(a1);
    const float sin_a2 = sin(a2);
    const float sin_a3 = sin(a3);
    const float cos_a1 = cos(a1);
    const float cos_a2 = cos(a2);
    const float cos_a3 = cos(a3);

    float R[9];
    R[0] = cos_a2 * cos_a3;
    R[1] = cos_a3 * sin_a1 * sin_a2 - cos_a1 * sin_a3;
    R[2] = sin_a1 * sin_a3 + cos_a1 * cos_a3 * sin_a2;
    R[3] = cos_a2 * sin_a3;
    R[4] = cos_a1 * cos_a3 + sin_a1 * sin_a2 * sin_a3;
    R[5] = cos_a1 * sin_a2 * sin_a3 - cos_a3 * sin_a1;
    R[6] = -sin_a2;
    R[7] = cos_a2 * sin_a1;
    R[8] = cos_a1 * cos_a2;

    float4 normal_perturbed;
    mat33_dot_vec3(R, normal, &normal_perturbed);

    if (vec3_dot_vec3(normal_perturbed, view_direction) >= 0.0f) {
        normal_perturbed = normal;
    }

    normalize_vec3(&normal_perturbed);
    return normal_perturbed;
}

__device__ void compute_homography(
    const Camera ref_camera,
    const Camera src_camera,
    const float4 plane_hypothesis,
    float* __restrict__ H) {
    float ref_C[3];
    float src_C[3];
    ref_C[0] =
        -(ref_camera.R[0] * ref_camera.t[0] + ref_camera.R[3] * ref_camera.t[1] +
          ref_camera.R[6] * ref_camera.t[2]);
    ref_C[1] =
        -(ref_camera.R[1] * ref_camera.t[0] + ref_camera.R[4] * ref_camera.t[1] +
          ref_camera.R[7] * ref_camera.t[2]);
    ref_C[2] =
        -(ref_camera.R[2] * ref_camera.t[0] + ref_camera.R[5] * ref_camera.t[1] +
          ref_camera.R[8] * ref_camera.t[2]);
    src_C[0] =
        -(src_camera.R[0] * src_camera.t[0] + src_camera.R[3] * src_camera.t[1] +
          src_camera.R[6] * src_camera.t[2]);
    src_C[1] =
        -(src_camera.R[1] * src_camera.t[0] + src_camera.R[4] * src_camera.t[1] +
          src_camera.R[7] * src_camera.t[2]);
    src_C[2] =
        -(src_camera.R[2] * src_camera.t[0] + src_camera.R[5] * src_camera.t[1] +
          src_camera.R[8] * src_camera.t[2]);

    float R_relative[9];
    float C_relative[3];
    float t_relative[3];
    R_relative[0] = src_camera.R[0] * ref_camera.R[0] + src_camera.R[1] * ref_camera.R[1] +
                    src_camera.R[2] * ref_camera.R[2];
    R_relative[1] = src_camera.R[0] * ref_camera.R[3] + src_camera.R[1] * ref_camera.R[4] +
                    src_camera.R[2] * ref_camera.R[5];
    R_relative[2] = src_camera.R[0] * ref_camera.R[6] + src_camera.R[1] * ref_camera.R[7] +
                    src_camera.R[2] * ref_camera.R[8];
    R_relative[3] = src_camera.R[3] * ref_camera.R[0] + src_camera.R[4] * ref_camera.R[1] +
                    src_camera.R[5] * ref_camera.R[2];
    R_relative[4] = src_camera.R[3] * ref_camera.R[3] + src_camera.R[4] * ref_camera.R[4] +
                    src_camera.R[5] * ref_camera.R[5];
    R_relative[5] = src_camera.R[3] * ref_camera.R[6] + src_camera.R[4] * ref_camera.R[7] +
                    src_camera.R[5] * ref_camera.R[8];
    R_relative[6] = src_camera.R[6] * ref_camera.R[0] + src_camera.R[7] * ref_camera.R[1] +
                    src_camera.R[8] * ref_camera.R[2];
    R_relative[7] = src_camera.R[6] * ref_camera.R[3] + src_camera.R[7] * ref_camera.R[4] +
                    src_camera.R[8] * ref_camera.R[5];
    R_relative[8] = src_camera.R[6] * ref_camera.R[6] + src_camera.R[7] * ref_camera.R[7] +
                    src_camera.R[8] * ref_camera.R[8];
    C_relative[0] = (ref_C[0] - src_C[0]);
    C_relative[1] = (ref_C[1] - src_C[1]);
    C_relative[2] = (ref_C[2] - src_C[2]);
    t_relative[0] = src_camera.R[0] * C_relative[0] + src_camera.R[1] * C_relative[1] +
                    src_camera.R[2] * C_relative[2];
    t_relative[1] = src_camera.R[3] * C_relative[0] + src_camera.R[4] * C_relative[1] +
                    src_camera.R[5] * C_relative[2];
    t_relative[2] = src_camera.R[6] * C_relative[0] + src_camera.R[7] * C_relative[1] +
                    src_camera.R[8] * C_relative[2];

    H[0] = R_relative[0] - t_relative[0] * plane_hypothesis.x / plane_hypothesis.w;
    H[1] = R_relative[1] - t_relative[0] * plane_hypothesis.y / plane_hypothesis.w;
    H[2] = R_relative[2] - t_relative[0] * plane_hypothesis.z / plane_hypothesis.w;
    H[3] = R_relative[3] - t_relative[1] * plane_hypothesis.x / plane_hypothesis.w;
    H[4] = R_relative[4] - t_relative[1] * plane_hypothesis.y / plane_hypothesis.w;
    H[5] = R_relative[5] - t_relative[1] * plane_hypothesis.z / plane_hypothesis.w;
    H[6] = R_relative[6] - t_relative[2] * plane_hypothesis.x / plane_hypothesis.w;
    H[7] = R_relative[7] - t_relative[2] * plane_hypothesis.y / plane_hypothesis.w;
    H[8] = R_relative[8] - t_relative[2] * plane_hypothesis.z / plane_hypothesis.w;

    float tmp[9];
    tmp[0] = H[0] / ref_camera.K[0];
    tmp[1] = H[1] / ref_camera.K[4];
    tmp[2] =
        -H[0] * ref_camera.K[2] / ref_camera.K[0] - H[1] * ref_camera.K[5] / ref_camera.K[4] + H[2];
    tmp[3] = H[3] / ref_camera.K[0];
    tmp[4] = H[4] / ref_camera.K[4];
    tmp[5] =
        -H[3] * ref_camera.K[2] / ref_camera.K[0] - H[4] * ref_camera.K[5] / ref_camera.K[4] + H[5];
    tmp[6] = H[6] / ref_camera.K[0];
    tmp[7] = H[7] / ref_camera.K[4];
    tmp[8] =
        -H[6] * ref_camera.K[2] / ref_camera.K[0] - H[7] * ref_camera.K[5] / ref_camera.K[4] + H[8];

    H[0] = src_camera.K[0] * tmp[0] + src_camera.K[2] * tmp[6];
    H[1] = src_camera.K[0] * tmp[1] + src_camera.K[2] * tmp[7];
    H[2] = src_camera.K[0] * tmp[2] + src_camera.K[2] * tmp[8];
    H[3] = src_camera.K[4] * tmp[3] + src_camera.K[5] * tmp[6];
    H[4] = src_camera.K[4] * tmp[4] + src_camera.K[5] * tmp[7];
    H[5] = src_camera.K[4] * tmp[5] + src_camera.K[5] * tmp[8];
    H[6] = src_camera.K[8] * tmp[6];
    H[7] = src_camera.K[8] * tmp[7];
    H[8] = src_camera.K[8] * tmp[8];
}

__device__ float2 compute_corresponding_point(const float* H, const int2 p) {
    float3 pt;
    pt.x = H[0] * p.x + H[1] * p.y + H[2];
    pt.y = H[3] * p.x + H[4] * p.y + H[5];
    pt.z = H[6] * p.x + H[7] * p.y + H[8];
    return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float4 transform_normal(const Camera camera, float4 plane_hypothesis) {
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[3] * plane_hypothesis.y +
                           camera.R[6] * plane_hypothesis.z;
    transformed_normal.y = camera.R[1] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y +
                           camera.R[7] * plane_hypothesis.z;
    transformed_normal.z = camera.R[2] * plane_hypothesis.x + camera.R[5] * plane_hypothesis.y +
                           camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}

__device__ float4 transform_normal_to_ref_cam(const Camera camera, float4 plane_hypothesis) {
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[1] * plane_hypothesis.y +
                           camera.R[2] * plane_hypothesis.z;
    transformed_normal.y = camera.R[3] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y +
                           camera.R[5] * plane_hypothesis.z;
    transformed_normal.z = camera.R[6] * plane_hypothesis.x + camera.R[7] * plane_hypothesis.y +
                           camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}

__device__ float4 compute_plane_hypothesis_from_normal_depth(
    const float* ref_depth_ptr, const float* ref_normal_ptr, const Camera camera, const int2 p) {
    float4 plane_hypothesis =
        make_float4(ref_normal_ptr[0], ref_normal_ptr[1], ref_normal_ptr[2], ref_depth_ptr[0]);
    // rotate the normal(from the world coordinate to the camera coordinate)
    plane_hypothesis = transform_normal_to_ref_cam(camera, plane_hypothesis);
    // get the distance to the coordinate origin from depth
    float depth = plane_hypothesis.w;
    plane_hypothesis.w = get_distance_to_origin(camera, p, depth, plane_hypothesis);
    return plane_hypothesis;
}

__device__ float compute_bilateral_weight(
    const float x_dist,
    const float y_dist,
    const float pix,
    const float center_pix,
    const float sigma_spatial,
    const float sigma_color) {
    // spatial_dist means the distance on the patch to the patch center
    const float spatial_dist = sqrt(x_dist * x_dist + y_dist * y_dist);
    const float color_dist = fabs(pix - center_pix);
    return exp(
        -spatial_dist / (2.0f * sigma_spatial * sigma_spatial) -
        color_dist / (2.0f * sigma_color * sigma_color));
}

__device__ float compute_bilateral_NCC(
    const float* __restrict__ ref_image,
    const Camera ref_camera,
    const float* __restrict__ src_image,
    const Camera src_camera,
    const int2 p,
    const float4 plane_hypothesis,
    const PatchMatchParams params) {
    const float cost_max = 2.0f;
    int32_t radius = params.patch_size / 2;

    // compute the homography matrix between two images via the camera
    // parameters
    float H[9];
    compute_homography(ref_camera, src_camera, plane_hypothesis, H);
    // get the corresponding pixel index on the src_image via homography matrix
    float2 pt = compute_corresponding_point(H, p);
    if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
        return cost_max;
    }

    const int32_t height = ref_camera.height;
    const int32_t width = ref_camera.width;
    float cost = 0.0f;
    {
        // init the immediate parameters
        float sum_ref = 0.0f;
        float sum_ref_ref = 0.0f;
        float sum_src = 0.0f;
        float sum_src_src = 0.0f;
        float sum_ref_src = 0.0f;
        float bilateral_weight_sum = 0.0f;
        const float ref_center_pix = ref_image[p.y * width + p.x];

        // patch looping
        for (int32_t i = -radius; i < radius + 1; i += params.radius_increment) {
            for (int32_t j = -radius; j < radius + 1; j += params.radius_increment) {
                // get the index and gray value on the ref_image
                const int2 ref_pt =
                    make_int2(clamp(p.x + i, 0, width - 1), clamp(p.y + j, 0, height - 1));
                const float ref_pix = ref_image[ref_pt.y * width + ref_pt.x];
                // get the index and gray value on the src_image
                float2 src_pt = compute_corresponding_point(H, ref_pt);
                const float src_pix = bilinear_interpolate(src_image, src_pt, height, width);

                /* get the bilateral weight, which describes the photometric
                   consistency between the reference and source patch
                 */
                float weight = compute_bilateral_weight(
                    i, j, ref_pix, ref_center_pix, params.sigma_spatial, params.sigma_color);

                sum_ref += weight * ref_pix;
                sum_ref_ref += weight * ref_pix * ref_pix;
                sum_src += weight * src_pix;
                sum_src_src += weight * src_pix * src_pix;
                sum_ref_src += weight * ref_pix * src_pix;
                bilateral_weight_sum += weight;
            }
        }
        // average the values of the patch
        const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
        sum_ref *= inv_bilateral_weight_sum;
        sum_ref_ref *= inv_bilateral_weight_sum;
        sum_src *= inv_bilateral_weight_sum;
        sum_src_src *= inv_bilateral_weight_sum;
        sum_ref_src *= inv_bilateral_weight_sum;

        // calculate the variance of the patch in ref_image and src_image
        const float var_ref = sum_ref_ref - sum_ref * sum_ref;
        const float var_src = sum_src_src - sum_src * sum_src;

        const float kMinVar = 1e-5f;
        if (var_ref < kMinVar || var_src < kMinVar) {  // for smooth area
            cost = cost_max;
        } else {
            // calculate the co-variance between ref_image and src_image
            // TODO: analyze here
            const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
            const float var_ref_src = sqrt(var_ref * var_src);
            cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
        }
        return cost;
    }
}

__device__ float compute_multi_view_initial_cost_and_selected_views(
    const float* __restrict__ ref_src_images,
    const Camera* __restrict__ ref_src_cameras,
    const int2 p,
    const float4 plane_hypothesis,
    uint32_t* __restrict__ selected_views,
    const PatchMatchParams params) {
    float cost_max = 2.0f;
    float cost_vector[32] = {2.0f};
    float cost_vector_copy[32] = {2.0f};
    int32_t cost_count = 0;
    int32_t num_valid_views = 0;

    const int32_t height = ref_src_cameras[0].height;
    const int32_t width = ref_src_cameras[0].width;

    for (int32_t i = 1; i < params.num_images; ++i) {
        // NCC cost
        float c = compute_bilateral_NCC(
            ref_src_images,
            ref_src_cameras[0],
            ref_src_images + i * height * width,
            ref_src_cameras[i],
            p,
            plane_hypothesis,
            params);
        cost_vector[i - 1] = c;
        cost_vector_copy[i - 1] = c;
        cost_count++;
        if (c < cost_max) {
            num_valid_views++;
        }
    }

    sort_small(cost_vector, cost_count);
    *selected_views = 0;

    // find the top_k match view and calculate the average cost
    int32_t top_k = min(num_valid_views, params.top_k);
    if (top_k > 0) {
        float cost = 0.0f;
        for (int32_t i = 0; i < top_k; ++i) {
            cost += cost_vector[i];
        }
        float cost_threshold = cost_vector[top_k - 1];
        for (int32_t i = 0; i < params.num_images - 1; ++i) {
            if (cost_vector_copy[i] <= cost_threshold) {
                set_bit(*selected_views, i);
            }
        }
        return cost / top_k;
    } else {
        return cost_max;
    }
}

__device__ void compute_multi_view_cost_vector(
    const float* __restrict__ ref_src_images,
    const Camera* __restrict__ ref_src_cameras,
    const int2 p,
    const float4 plane_hypothesis,
    float* __restrict__ cost_vector,
    const PatchMatchParams params) {
    const int32_t height = ref_src_cameras[0].height;
    const int32_t width = ref_src_cameras[0].width;
    for (int32_t i = 1; i < params.num_images; ++i) {
        cost_vector[i - 1] = compute_bilateral_NCC(
            ref_src_images,
            ref_src_cameras[0],
            ref_src_images + i * height * width,
            ref_src_cameras[i],
            p,
            plane_hypothesis,
            params);
    }
}

__device__ float3
get_3D_point_on_World(const float x, const float y, const float depth, const Camera camera) {
    float3 pointX;
    float3 tmpX;
    // Reprojection
    pointX.x = depth * (x - camera.K[2]) / camera.K[0];
    pointX.y = depth * (y - camera.K[5]) / camera.K[4];
    pointX.z = depth;

    // Rotation
    tmpX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z;
    tmpX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z;
    tmpX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z;

    // Transformation
    float3 C;
    C.x = -(camera.R[0] * camera.t[0] + camera.R[3] * camera.t[1] + camera.R[6] * camera.t[2]);
    C.y = -(camera.R[1] * camera.t[0] + camera.R[4] * camera.t[1] + camera.R[7] * camera.t[2]);
    C.z = -(camera.R[2] * camera.t[0] + camera.R[5] * camera.t[1] + camera.R[8] * camera.t[2]);
    pointX.x = tmpX.x + C.x;
    pointX.y = tmpX.y + C.y;
    pointX.z = tmpX.z + C.z;

    return pointX;
}

__device__ void project_on_camera(
    const float3 PointX, const Camera camera, float2& point, float& depth) {
    float3 tmp;
    tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
    tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
    tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

    depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
    point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
    point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;
}

__device__ float compute_geom_consistency_cost(
    const float* __restrict__ src_depths,
    const Camera ref_camera,
    const Camera src_camera,
    const float4 plane_hypothesis,
    const int2 p) {
    const float max_cost = 5.0f;

    float depth = compute_depth_from_plane_hypothesis(ref_camera, plane_hypothesis, p);
    float3 forward_point = get_3D_point_on_World(p.x, p.y, depth, ref_camera);

    float2 src_pt;
    float src_d;
    project_on_camera(forward_point, src_camera, src_pt, src_d);
    const int32_t height = ref_camera.height;
    const int32_t width = ref_camera.width;
    const float src_depth = bilinear_interpolate(src_depths, src_pt, height, width);

    if (src_depth == 0.0f) {
        return max_cost;
    }

    float3 src_3D_pt = get_3D_point_on_World(src_pt.x, src_pt.y, src_depth, src_camera);

    float2 backward_point;
    float ref_d;
    project_on_camera(src_3D_pt, ref_camera, backward_point, ref_d);

    const float diff_col = p.x - backward_point.x;
    const float diff_row = p.y - backward_point.y;
    return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

__device__ void plane_hypothesis_refinement(
    const float* __restrict__ ref_src_images,
    float* __restrict__ ref_src_depths,
    const Camera* __restrict__ ref_src_cameras,
    float4* __restrict__ plane_hypothesis,
    float* __restrict__ depth,
    float* __restrict__ cost,
    hiprandState* __restrict__ rand_state,
    const float* __restrict__ view_weights,
    const float weight_norm,
    const int2 p,
    const PatchMatchParams params) {
    float perturbation = 0.02f;

    float depth_sigma = (params.depth_max - params.depth_min) / 64.0f;
    float two_depth_sigma_squared = 2 * depth_sigma * depth_sigma;
    float angle_sigma = M_PI * (5.0f / 180.0f);
    float two_angle_sigma_squared = 2 * angle_sigma * angle_sigma;

    float depth_rand;
    float4 plane_hypothesis_rand;

    const Camera ref_camera = ref_src_cameras[0];
    const int32_t height = ref_camera.height;
    const int32_t width = ref_camera.width;

    depth_rand =
        hiprand_uniform(rand_state) * (params.depth_max - params.depth_min) + params.depth_min;
    plane_hypothesis_rand = generate_random_normal(ref_camera, p, rand_state, *depth);
    float depth_perturbed = *depth;
    const float depth_min_perturbed = (1 - perturbation) * depth_perturbed;
    const float depth_max_perturbed = (1 + perturbation) * depth_perturbed;
    do {
        depth_perturbed = hiprand_uniform(rand_state) * (depth_max_perturbed - depth_min_perturbed) +
                          depth_min_perturbed;
    } while (depth_perturbed < params.depth_min && depth_perturbed > params.depth_max);
    float4 plane_hypothesis_perturbed = generate_perturbed_normal(
        ref_camera, p, *plane_hypothesis, rand_state, perturbation * M_PI);

    const int32_t num_planes = 5;
    float depth_candidates[num_planes] = {depth_rand, *depth, depth_rand, *depth, depth_perturbed};
    float4 normal_candidates[num_planes] = {*plane_hypothesis,
                                            plane_hypothesis_rand,
                                            plane_hypothesis_rand,
                                            plane_hypothesis_perturbed,
                                            *plane_hypothesis};

    for (int32_t i = 0; i < num_planes; ++i) {
        float cost_vector[32] = {2.0f};
        float4 temp_plane_hypothesis = normal_candidates[i];
        temp_plane_hypothesis.w = get_distance_to_origin(
            ref_camera,
            p,
            depth_candidates[i],
            temp_plane_hypothesis);  // dists[i];
        compute_multi_view_cost_vector(
            ref_src_images, ref_src_cameras, p, temp_plane_hypothesis, cost_vector, params);

        float temp_cost = 0.0f;
        for (int32_t j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                temp_cost += view_weights[j] *
                             (cost_vector[j] + 0.1f * compute_geom_consistency_cost(
                                                          ref_src_depths + (j + 1) * height * width,
                                                          ref_camera,
                                                          ref_src_cameras[j + 1],
                                                          temp_plane_hypothesis,
                                                          p));
            }
        }
        temp_cost /= weight_norm;

        float depth_before =
            compute_depth_from_plane_hypothesis(ref_camera, temp_plane_hypothesis, p);
        if (depth_before >= params.depth_min && depth_before <= params.depth_max &&
            temp_cost < *cost) {
            *depth = depth_before;
            *plane_hypothesis = temp_plane_hypothesis;
            *cost = temp_cost;
        }
    }
}

__device__ void checkerboard_propagation(
    const float* __restrict__ ref_src_images,
    float* __restrict__ ref_src_depths,
    const Camera* __restrict__ cameras,
    float4* __restrict__ plane_hypotheses,
    float* __restrict__ costs,
    hiprandState* __restrict__ rand_states,
    uint32_t* __restrict__ selected_views,
    float* __restrict__ view_weights,
    float* __restrict__ weight_norm,
    float* __restrict__ depth_now,
    const int2 p,
    const PatchMatchParams params,
    const int32_t iter) {
    /* get the pixel and query indices
                o u_f
                |
                o u_n
      l_f  l_n  |  r_n  r_f
        o---o---c---o---o
                |
                o d_n
                |
                o d_f
     */
    const Camera ref_camera = cameras[0];
    const int32_t height = ref_camera.height;
    const int32_t width = ref_camera.width;
    const int32_t center = p.y * width + p.x;
    int32_t left_near = center - 1;
    int32_t left_far = center - 3;
    int32_t right_near = center + 1;
    int32_t right_far = center + 3;
    int32_t up_near = center - width;
    int32_t up_far = center - 3 * width;
    int32_t down_near = center + width;
    int32_t down_far = center + 3 * width;

    // Adaptive Checkerboard Sampling
    float cost_array[8][32] = {2.0f};
    // 0 -- up_near, 1 -- up_far, 2 -- down_near, 3 -- down_far, 4 -- left_near,
    // 5 -- left_far, 6 -- right_near, 7 -- right_far
    bool flag[8] = {false};

    float cost_min;
    int32_t cost_min_point;

    // up_far
    if (p.y > 2) {  // out of bound judgement
        flag[1] = true;
        cost_min = costs[up_far];
        cost_min_point = up_far;
        // query the minimum cost along the vertical direction
        for (int32_t i = 1; i < 11; ++i) {
            if (p.y > 2 + 2 * i) {
                int32_t point_temp = up_far - 2 * i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        up_far = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[up_far], cost_array[1], params);
    }

    // dwon_far
    if (p.y < height - 3) {  // out of bound judgement
        flag[3] = true;
        cost_min = costs[down_far];
        cost_min_point = down_far;
        // query the minimum cost along the vertical direction
        for (int32_t i = 1; i < 11; ++i) {
            if (p.y < height - 3 - 2 * i) {
                int32_t point_temp = down_far + 2 * i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        down_far = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[down_far], cost_array[3], params);
    }

    // left_far
    if (p.x > 2) {  // out of bound judgement
        flag[5] = true;
        cost_min = costs[left_far];
        cost_min_point = left_far;
        // query the minimum cost along the horizontal direction
        for (int32_t i = 1; i < 11; ++i) {
            if (p.x > 2 + 2 * i) {
                int32_t point_temp = left_far - 2 * i;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        left_far = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[left_far], cost_array[5], params);
    }

    // right_far
    if (p.x < width - 3) {  // out of bound judgement
        flag[7] = true;
        cost_min = costs[right_far];
        cost_min_point = right_far;
        // query the minimum cost along the horizontal direction
        for (int32_t i = 1; i < 11; ++i) {
            if (p.x < width - 3 - 2 * i) {
                int32_t point_temp = right_far + 2 * i;
                if (cost_min < costs[point_temp]) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        right_far = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[right_far], cost_array[7], params);
    }

    // up_near
    if (p.y > 0) {  // out of bound judgement
        flag[0] = true;
        cost_min = costs[up_near];
        cost_min_point = up_near;
        // query the minimum cost along the v-type direction
        for (int32_t i = 0; i < 3; ++i) {
            if (p.y > 1 + i && p.x > i) {
                int32_t point_temp = up_near - (1 + i) * width - i;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
            if (p.y > 1 + i && p.x < width - 1 - i) {
                int32_t point_temp = up_near - (1 + i) * width + i;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        up_near = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[up_near], cost_array[0], params);
    }

    // down_near
    if (p.y < height - 1) {  // out of bound judgement
        flag[2] = true;
        cost_min = costs[down_near];
        cost_min_point = down_near;
        // query the minimum cost along the v-type direction
        for (int32_t i = 0; i < 3; ++i) {
            if (p.y < height - 2 - i && p.x > i) {
                int32_t point_temp = down_near + (1 + i) * width - i;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
            if (p.y < height - 2 - i && p.x < width - 1 - i) {
                int32_t point_temp = down_near + (1 + i) * width + i;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        down_near = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[down_near], cost_array[2], params);
    }

    // left_near
    if (p.x > 0) {  // out of bound judgement
        flag[4] = true;
        cost_min = costs[left_near];
        cost_min_point = left_near;
        // query the minimum cost along the v-type direction
        for (int32_t i = 0; i < 3; ++i) {
            if (p.x > 1 + i && p.y > i) {
                int32_t point_temp = left_near - (1 + i) - i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
            if (p.x > 1 + i && p.y < height - 1 - i) {
                int32_t point_temp = left_near - (1 + i) + i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        left_near = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[left_near], cost_array[4], params);
    }

    // right_near
    if (p.x < width - 1) {  // out of bound judgement
        flag[6] = true;
        cost_min = costs[right_near];
        cost_min_point = right_near;
        // query the minimum cost along the v-type direction
        for (int32_t i = 0; i < 3; ++i) {
            if (p.x < width - 2 - i && p.y > i) {
                int32_t point_temp = right_near + (1 + i) - i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
            if (p.x < width - 2 - i && p.y < height - 1 - i) {
                int32_t point_temp = right_near + (1 + i) + i * width;
                if (costs[point_temp] < cost_min) {
                    cost_min = costs[point_temp];
                    cost_min_point = point_temp;
                }
            }
        }
        right_near = cost_min_point;
        compute_multi_view_cost_vector(
            ref_src_images, cameras, p, plane_hypotheses[right_near], cost_array[6], params);
    }
    const int32_t positions[8] = {
        up_near, up_far, down_near, down_far, left_near, left_far, right_near, right_far};

    // Multi-hypothesis Joint View Selection
    // float view_weights[32] = {0.0f};
    float view_selection_priors[32] = {0.0f};
    int32_t neighbor_positions[4] = {center - width, center + width, center - 1, center + 1};
    for (int32_t i = 0; i < 4; ++i) {
        if (flag[2 * i]) {  // query the 4-nearest pixel
            // accumulate the view selection score
            for (int32_t j = 0; j < params.num_images - 1; ++j) {
                if (is_set(selected_views[neighbor_positions[i]], j) == 1) {
                    view_selection_priors[j] += 0.9f;
                } else {
                    view_selection_priors[j] += 0.1f;
                }
            }
        }
    }

    // calculate the PDF
    float sampling_probs[32] = {0.0f};
    float cost_threshold = 0.8 * expf((iter) * (iter) / (-90.0f));
    for (int32_t i = 0; i < params.num_images - 1; i++) {
        float count = 0;
        int32_t count_false = 0;
        float tmpw = 0;
        // statistic the cost_array for checkerboard
        for (int32_t j = 0; j < 8; j++) {
            if (cost_array[j][i] < cost_threshold) {
                tmpw += expf(cost_array[j][i] * cost_array[j][i] / (-0.18f));
                count++;
            }
            if (cost_array[j][i] > 1.2f) {
                count_false++;
            }
        }
        if (count > 2 && count_false < 3) {
            sampling_probs[i] = tmpw / count;
        } else if (count_false < 3) {
            sampling_probs[i] = expf(cost_threshold * cost_threshold / (-0.32f));
        }
        sampling_probs[i] = sampling_probs[i] * view_selection_priors[i];
    }

    // PDF to CDF
    transform_PDF_to_CDF(sampling_probs, params.num_images - 1);
    for (int32_t sample = 0; sample < 15; ++sample) {
        const float rand_prob = hiprand_uniform(&rand_states[center]) - FLT_EPSILON;

        for (int32_t image_id = 0; image_id < params.num_images - 1; ++image_id) {
            const float prob = sampling_probs[image_id];
            if (prob > rand_prob) {
                view_weights[image_id] += 1.0f;
                break;
            }
        }
    }

    // store the selected views and their weights
    uint32_t temp_selected_views = 0;
    int32_t num_selected_view = 0;
    *weight_norm = 0;
    for (int32_t i = 0; i < params.num_images - 1; ++i) {
        if (view_weights[i] > 0) {
            set_bit(temp_selected_views, i);
            *weight_norm += view_weights[i];
            num_selected_view++;
        }
    }

    // calculate the final cost for 8 query pixels
    float final_costs[8] = {0.0f};
    for (int32_t i = 0; i < 8; ++i) {
        for (int32_t j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                if (flag[i]) {
                    final_costs[i] +=
                        view_weights[j] *
                        (cost_array[i][j] + 0.1f * compute_geom_consistency_cost(
                                                       ref_src_depths + (j + 1) * height * width,
                                                       ref_camera,
                                                       cameras[j + 1],
                                                       plane_hypotheses[positions[i]],
                                                       p));
                } else {
                    final_costs[i] += view_weights[j] * cost_array[i][j];
                }
            }
        }
        final_costs[i] /= *weight_norm;
    }

    // find the index of the cost minimum
    const int32_t min_cost_idx = find_min_cost_index(final_costs, 8);

    // find the cost of the current pixel
    float cost_vector_now[32] = {2.0f};
    compute_multi_view_cost_vector(
        ref_src_images, cameras, p, plane_hypotheses[center], cost_vector_now, params);
    float cost_now = 0.0f;
    for (int32_t i = 0; i < params.num_images - 1; ++i) {
        cost_now += view_weights[i] *
                    (cost_vector_now[i] + 0.1f * compute_geom_consistency_cost(
                                                     ref_src_depths + (i + 1) * height * width,
                                                     ref_camera,
                                                     cameras[i + 1],
                                                     plane_hypotheses[center],
                                                     p));
    }
    // get the average of cost as the cost of the current pixel
    cost_now /= *weight_norm;
    costs[center] = cost_now;
    // get the depth according to the plane hypothesis
    *depth_now = compute_depth_from_plane_hypothesis(ref_camera, plane_hypotheses[center], p);

    if (flag[min_cost_idx]) {
        const int32_t pos = positions[min_cost_idx];
        float depth_before =
            compute_depth_from_plane_hypothesis(ref_camera, plane_hypotheses[pos], p);

        if (depth_before >= params.depth_min && depth_before <= params.depth_max &&
            final_costs[min_cost_idx] < cost_now) {
            *depth_now = depth_before;
            plane_hypotheses[center] = plane_hypotheses[pos];
            costs[center] = final_costs[min_cost_idx];
            selected_views[center] = temp_selected_views;
        }
    }
}

// kernels

// __launch_bounds__(CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void initialization(
    const float* __restrict__ ref_src_images,
    const float* __restrict__ ref_src_depths,
    const float* __restrict__ ref_src_normals,
    Camera* __restrict__ cameras_cuda,      // num_images
    float4* __restrict__ plane_hypotheses,  // [H, W]
    float* __restrict__ costs,              // [H, W]
    hiprandState* __restrict__ rand_states,  // [H, W]
    uint32_t* __restrict__ selected_views,  // [H, W]
    const PatchMatchParams params) {
    // the pixel index
    const int2 p =
        make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    int32_t width = cameras_cuda[0].width;
    int32_t height = cameras_cuda[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    // 1d index
    const int32_t center = p.y * width + p.x;
    // hiprand_init(clock64(), p.y, p.x, &rand_states[center]);
    hiprand_init(RANDOM_SEED, p.y, p.x, &rand_states[center]);

    // get the plane hypothesis for the current pixel
    const int32_t img_offset = p.y * width + p.x;

    float4 plane_hypothesis = compute_plane_hypothesis_from_normal_depth(
        ref_src_depths + img_offset, ref_src_normals + img_offset * 3, cameras_cuda[0], p);

    // rewrite the plane hypothesis for the current pixel
    plane_hypotheses[center] = plane_hypothesis;
    // calculate the cost of the current pixel via NCC
    costs[center] = compute_multi_view_initial_cost_and_selected_views(
        ref_src_images, cameras_cuda, p, plane_hypotheses[center], &selected_views[center], params);
}

// __launch_bounds__(CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void black_red_pixel_update(
    const float* __restrict__ ref_src_images,
    float* __restrict__ ref_src_depths,
    Camera* __restrict__ ref_src_cameras,   // num_images
    float4* __restrict__ plane_hypotheses,  // [H, W]
    float* __restrict__ costs,              // [H, W]
    hiprandState* __restrict__ rand_states,  // [H, W]
    uint32_t* __restrict__ selected_views,  // [H, W]
    const PatchMatchParams params,
    const int32_t iter,
    const bool black) {
    // get the pixel index for black
    int2 p =
        make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = black ? p.y * 2 : p.y * 2 + 1;
    } else {
        p.y = black ? p.y * 2 + 1 : p.y * 2;
    }
    int32_t width = ref_src_cameras[0].width;
    int32_t height = ref_src_cameras[0].height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    // perform checkerboard propagation
    float view_weights[32] = {0.0f};
    float weight_norm;
    float depth_now;
    checkerboard_propagation(
        ref_src_images,
        ref_src_depths,
        ref_src_cameras,
        plane_hypotheses,  // output
        costs,             // output
        rand_states,
        selected_views,  // output
        view_weights,    // output
        &weight_norm,    // output
        &depth_now,      // output
        p,
        params,
        iter);

    const int32_t center = p.y * width + p.x;
    plane_hypothesis_refinement(
        ref_src_images,
        ref_src_depths,
        ref_src_cameras,
        &plane_hypotheses[center],  // output
        &depth_now,                 // output
        &costs[center],             // output
        &rand_states[center],
        view_weights,
        weight_norm,
        p,
        params);
}

// __launch_bounds__(CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void get_depth_and_normal(
    Camera* __restrict__ cameras,
    float4* __restrict__ plane_hypotheses,
    const PatchMatchParams params) {
    const int2 p =
        make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int32_t width = cameras[0].width;
    const int32_t height = cameras[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    const int32_t center = p.y * width + p.x;
    plane_hypotheses[center].w =
        compute_depth_from_plane_hypothesis(cameras[0], plane_hypotheses[center], p);
    plane_hypotheses[center] = transform_normal(cameras[0], plane_hypotheses[center]);
}

// __launch_bounds__(CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void black_red_pixel_filter(
    const Camera* __restrict__ ref_src_cameras,
    float4* __restrict__ plane_hypotheses,
    float* __restrict__ costs,
    const bool black) {
    int2 p =
        make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = black ? p.y * 2 : p.y * 2 + 1;
    } else {
        p.y = black ? p.y * 2 + 1 : p.y * 2;
    }

    const Camera ref_camera = ref_src_cameras[0];
    const int32_t width = ref_camera.width;
    const int32_t height = ref_camera.height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    const int32_t center = p.y * width + p.x;

    float filter[21];
    int32_t index = 0;

    filter[index++] = plane_hypotheses[center].w;

    // Left
    const int32_t left = center - 1;
    const int32_t leftleft = center - 3;

    // Up
    const int32_t up = center - width;
    const int32_t upup = center - 3 * width;

    // Down
    const int32_t down = center + width;
    const int32_t downdown = center + 3 * width;

    // Right
    const int32_t right = center + 1;
    const int32_t rightright = center + 3;

    if (costs[center] < 0.001f) {
        return;
    }

    if (p.y > 0) {
        filter[index++] = plane_hypotheses[up].w;
    }
    if (p.y > 2) {
        filter[index++] = plane_hypotheses[upup].w;
    }
    if (p.y > 4) {
        filter[index++] = plane_hypotheses[upup - width * 2].w;
    }
    if (p.y < height - 1) {
        filter[index++] = plane_hypotheses[down].w;
    }
    if (p.y < height - 3) {
        filter[index++] = plane_hypotheses[downdown].w;
    }
    if (p.y < height - 5) {
        filter[index++] = plane_hypotheses[downdown + width * 2].w;
    }
    if (p.x > 0) {
        filter[index++] = plane_hypotheses[left].w;
    }
    if (p.x > 2) {
        filter[index++] = plane_hypotheses[leftleft].w;
    }
    if (p.x > 4) {
        filter[index++] = plane_hypotheses[leftleft - 2].w;
    }
    if (p.x < width - 1) {
        filter[index++] = plane_hypotheses[right].w;
    }
    if (p.x < width - 3) {
        filter[index++] = plane_hypotheses[rightright].w;
    }
    if (p.x < width - 5) {
        filter[index++] = plane_hypotheses[rightright + 2].w;
    }
    if (p.y > 0 && p.x < width - 2) {
        filter[index++] = plane_hypotheses[up + 2].w;
    }
    if (p.y < height - 1 && p.x < width - 2) {
        filter[index++] = plane_hypotheses[down + 2].w;
    }
    if (p.y > 0 && p.x > 1) {
        filter[index++] = plane_hypotheses[up - 2].w;
    }
    if (p.y < height - 1 && p.x > 1) {
        filter[index++] = plane_hypotheses[down - 2].w;
    }
    if (p.x > 0 && p.y > 2) {
        filter[index++] = plane_hypotheses[left - width * 2].w;
    }
    if (p.x < width - 1 && p.y > 2) {
        filter[index++] = plane_hypotheses[right - width * 2].w;
    }
    if (p.x > 0 && p.y < height - 2) {
        filter[index++] = plane_hypotheses[left + width * 2].w;
    }
    if (p.x < width - 1 && p.y < height - 2) {
        filter[index++] = plane_hypotheses[right + width * 2].w;
    }

    // average the depth
    sort_small(filter, index);
    int32_t median_index = index / 2;
    if (index % 2 == 0) {
        plane_hypotheses[center].w = (filter[median_index - 1] + filter[median_index]) / 2;
    } else {
        plane_hypotheses[center].w = filter[median_index];
    }
}

// __launch_bounds__(CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void export_plane_hypotheses_cost(
    const float4* __restrict__ plane_ptr,  // [H, W]
    const float* __restrict__ cost_ptr,    // [H, W]
    const int32_t height,
    const int32_t width,
    // output
    float* __restrict__ output_depth_ptr,   // [H, W]
    float* __restrict__ output_normal_ptr,  // [H, W, 3]
    float* __restrict__ output_cost_ptr     // [H, W]
) {
    // const int32_t row = blockIdx.x;
    // const int32_t col = threadIdx.x;

    // const int32_t index = row * width + col;

    const int32_t pixel_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixel_id >= height * width){
        return;
    } 
    const int32_t row = pixel_id / width;
    const int32_t col = pixel_id % width;
    
    const int32_t index = row * width + col;

    output_depth_ptr[index] = plane_ptr[index].w;
    output_normal_ptr[index * 3 + 0] = plane_ptr[index].x;
    output_normal_ptr[index * 3 + 1] = plane_ptr[index].y;
    output_normal_ptr[index * 3 + 2] = plane_ptr[index].z;
    output_cost_ptr[index] = cost_ptr[index];
}

mvs::PatchMatcher::PatchMatcher() {
    num_images = 0;
    params = PatchMatchParams();
}
mvs::PatchMatcher::~PatchMatcher() {
}

void mvs::PatchMatcher::add_samples(
    const vector<Problem> problems,
    const vector<Camera> cameras,
    const Tensor& images,
    const Tensor& depths,
    const Tensor& normals,
    const Tensor& costs) {
    // check torch tensor
    CHECK_CPU_INPUT(images);
    CHECK_CPU_INPUT(depths);
    CHECK_CPU_INPUT(normals);
    CHECK_CPU_INPUT(costs);

    // get the number of images
    num_images = problems.size();
    cameras_host = cameras;
    problems_host = problems;

    height = cameras[0].height;
    width = cameras[0].width;

    // allocate to copy cuda
    images_host = images;
    depths_host = depths;
    normals_host = normals;
    costs_host = costs;
    CUDA_CHECK_THROW(hipDeviceSynchronize());
}

// launch function

tuple<Tensor, Tensor, Tensor> mvs::PatchMatcher::run_patch_match(
    const int32_t problem_idx, const bool verbose) {
    // define the grid and block for cuda parallelization
    dim3 grid_size_randinit;
    grid_size_randinit.x = (width + 16 - 1) / 16;
    grid_size_randinit.y = (height + 16 - 1) / 16;
    grid_size_randinit.z = 1;
    dim3 block_size_randinit;
    block_size_randinit.x = 16;
    block_size_randinit.y = 16;
    block_size_randinit.z = 1;

    // half pixels
    dim3 grid_size_checkerboard;
    grid_size_checkerboard.x = (width + BLOCK_W - 1) / BLOCK_W;
    grid_size_checkerboard.y = ((height / 2) + BLOCK_H - 1) / BLOCK_H;
    grid_size_checkerboard.z = 1;
    dim3 block_size_checkerboard;
    block_size_checkerboard.x = BLOCK_W;
    block_size_checkerboard.y = BLOCK_H;
    block_size_checkerboard.z = 1;

    // set parameters
    const Problem problem_host = problems_host[problem_idx];
    params.num_images = problem_host.num_ngb + 1;
    const int32_t ref_id = problem_host.ref_image_id;
    params.depth_min = cameras_host[ref_id].depth_min * 0.6;
    params.depth_max = cameras_host[ref_id].depth_max * 1.2;

    // cuda init
    float4* plane_hypotheses_cuda = NULL;
    hipMalloc((void**)&plane_hypotheses_cuda, sizeof(float4) * (height * width));
    hiprandState* rand_states_cuda = NULL;
    hipMalloc((void**)&rand_states_cuda, sizeof(hiprandState) * (height * width));
    uint32_t* selected_views_cuda = NULL;
    hipMalloc((void**)&selected_views_cuda, sizeof(uint32_t) * (height * width));

    Tensor cost_cuda = costs_host.index({ref_id}).to(torch::kCUDA);
    Tensor images_cuda = torch::zeros(
        {params.num_images, height, width},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor depths_cuda = torch::zeros(
        {params.num_images, height, width},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor normals_cuda = torch::zeros(
        {params.num_images, height, width, 3},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Camera* cameras_cuda = NULL;
    {
        images_cuda.index_put_({0}, images_host.index({ref_id}).to(torch::kCUDA));
        depths_cuda.index_put_({0}, depths_host.index({ref_id}).to(torch::kCUDA));
        normals_cuda.index_put_({0}, normals_host.index({ref_id}).to(torch::kCUDA));

        const int32_t* src_ids = problem_host.src_image_ids;
        vector<Camera> ref_src_cameras;
        ref_src_cameras.push_back(cameras_host[ref_id]);
        for (uint16_t i = 0; i < problem_host.num_ngb; ++i) {
            const int32_t src_id = src_ids[i];
            images_cuda.index_put_({i + 1}, images_host.index({src_id}).to(torch::kCUDA));
            depths_cuda.index_put_({i + 1}, depths_host.index({src_id}).to(torch::kCUDA));
            normals_cuda.index_put_({i + 1}, normals_host.index({src_id}).to(torch::kCUDA));
            ref_src_cameras.push_back(cameras_host[src_id]);
        }
        hipMalloc((void**)&cameras_cuda, sizeof(Camera) * params.num_images);
        hipMemcpy(
            cameras_cuda,
            &ref_src_cameras[0],
            sizeof(Camera) * params.num_images,
            hipMemcpyHostToDevice);
    }
    CUDA_CHECK_THROW(hipDeviceSynchronize());

    initialization<<<grid_size_randinit, block_size_randinit>>>(
        images_cuda.data_ptr<float>(),
        depths_cuda.data_ptr<float>(),
        normals_cuda.data_ptr<float>(),
        cameras_cuda,                 // num_images
        plane_hypotheses_cuda,        // [H, W]
        cost_cuda.data_ptr<float>(),  // [H, W]
        rand_states_cuda,             // [H, W]
        selected_views_cuda,          // [H, W]
        params);
    CUDA_CHECK_THROW(hipDeviceSynchronize());

    for (int32_t i = 0; i < params.max_iterations; ++i) {
        // black pixel update
        black_red_pixel_update<<<grid_size_checkerboard, block_size_checkerboard>>>(
            images_cuda.data_ptr<float>(),
            depths_cuda.data_ptr<float>(),
            cameras_cuda,                 // num_images
            plane_hypotheses_cuda,        // [H, W]
            cost_cuda.data_ptr<float>(),  // [H, W]
            rand_states_cuda,             // [H, W]
            selected_views_cuda,          // [H, W]
            params,
            i,
            true);
        CUDA_CHECK_THROW(hipDeviceSynchronize());
        // red pixel update
        black_red_pixel_update<<<grid_size_checkerboard, block_size_checkerboard>>>(
            images_cuda.data_ptr<float>(),
            depths_cuda.data_ptr<float>(),
            cameras_cuda,                 // num_images
            plane_hypotheses_cuda,        // [H, W]
            cost_cuda.data_ptr<float>(),  // [H, W]
            rand_states_cuda,             // [H, W]
            selected_views_cuda,          // [H, W]
            params,
            i,
            false);
        CUDA_CHECK_THROW(hipDeviceSynchronize());
        if (verbose) {
            std::cout << "iteration: " << i << std::endl;
        }
    }

    // get the depth and normal in the world coordinate
    get_depth_and_normal<<<grid_size_randinit, block_size_randinit>>>(
        cameras_cuda, plane_hypotheses_cuda, params);
    CUDA_CHECK_THROW(hipDeviceSynchronize());

    // average(smooth) the depth according to the black-red checkerboard
    // black pixel filter
    black_red_pixel_filter<<<grid_size_checkerboard, block_size_checkerboard>>>(
        cameras_cuda, plane_hypotheses_cuda, cost_cuda.data_ptr<float>(), true);
    CUDA_CHECK_THROW(hipDeviceSynchronize());
    // red pixel filter
    black_red_pixel_filter<<<grid_size_checkerboard, block_size_checkerboard>>>(
        cameras_cuda, plane_hypotheses_cuda, cost_cuda.data_ptr<float>(), false);
    CUDA_CHECK_THROW(hipDeviceSynchronize());

    Tensor output_depths = torch::zeros(
        {height, width}, torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor output_normals = torch::zeros(
        {height, width, 3}, torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor output_costs = torch::zeros(
        {height, width}, torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    CUDA_CHECK_THROW(hipDeviceSynchronize());

    // export_plane_hypotheses_cost<<<height, width>>>(
    //     plane_hypotheses_cuda,
    //     cost_cuda.data_ptr<float>(),  // [H, W]
    //     height,
    //     width,
    //     // output
    //     output_depths.data_ptr<float>(),
    //     output_normals.data_ptr<float>(),
    //     output_costs.data_ptr<float>());
    const int32_t num_threads = 256;
    const int32_t num_blocks = (height * width - 1) / num_threads + 1;
    export_plane_hypotheses_cost<<<num_blocks, num_threads>>>(
        plane_hypotheses_cuda,
        cost_cuda.data_ptr<float>(),  // [H, W]
        height,
        width,
        // output
        output_depths.data_ptr<float>(),
        output_normals.data_ptr<float>(),
        output_costs.data_ptr<float>());

    hipFree(cameras_cuda);
    hipFree(plane_hypotheses_cuda);
    hipFree(rand_states_cuda);
    hipFree(selected_views_cuda);

    return std::make_tuple(output_depths, output_normals, output_costs);
}